#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>

const int warp_size = 32;
const unsigned int full_mask = 0xffffffff;

__inline__ __device__ int warpReduce(int val) {
  for (int i = warp_size / 2; i > 0; i /= 2) {
    val += __shfl_down_sync(full_mask, val, i);
  }
  return val;
}

__inline__ __device__ int threadSum(int *d_arr, size_t length,
                                    size_t start_offset, size_t stride) {
  int sum = 0;
  for (size_t i = start_offset; i < length; i += stride) {
    sum += d_arr[i];
  }
  return sum;
}

__global__ void reduce_with_buffer(int *array, size_t length, int *buffer) {
  __shared__ int temp[warp_size];
  const size_t global_id = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t global_dim = blockDim.x * gridDim.x;

  int thread_sum = threadSum(array, length, global_id, global_dim);
  thread_sum = warpReduce(thread_sum);
  const int lane_id = threadIdx.x & 31;
  if (lane_id == 0) {
    const int warp_id = threadIdx.x / warp_size;
    temp[warp_id] = thread_sum;
  }
  // __syncthreads();
  __threadfence_block();

  const int numWarps = blockDim.x / warp_size;
  if (threadIdx.x < warp_size) {
    // thread_sum = threadIdx.x < numWarps ? temp[threadIdx.x] : 0.0f;
    thread_sum = threadSum(temp, numWarps, threadIdx.x, warp_size);
    thread_sum = warpReduce(thread_sum);
    if (threadIdx.x == 0) {
      buffer[blockIdx.x] = thread_sum;
    }
  }
  __syncthreads();

  if (blockIdx.x == 0 && threadIdx.x < warp_size) {
    thread_sum = threadSum(buffer, gridDim.x, threadIdx.x, warp_size);
    thread_sum = warpReduce(thread_sum);
    if (threadIdx.x == 0) {
      buffer[0] = thread_sum;
    }
  }
}

int reduce_gpu(int *arr, size_t length) {
  int *d_arr, *d_buffer;
  hipMalloc(&d_arr, sizeof(int) * length);
  hipMemcpyAsync(d_arr, arr, length * sizeof(int), hipMemcpyDefault);
  const int numBlocks = 32;
  const int numThreads = 128;
  hipMalloc(&d_buffer, sizeof(int) * numBlocks);
  reduce_with_buffer<<<numBlocks, numThreads>>>(d_arr, length, d_buffer);
  hipStreamSynchronize(0);
  int result = 0;
  hipMemcpy(&result, d_buffer, sizeof(int), hipMemcpyDefault);
  hipFree(d_arr);
  hipFree(d_buffer);
  return result;
}

float reduce_cub_float(float *arr, size_t length) {
  float *d_temp{nullptr}, *d_in, *d_out;
  size_t temp_bytes = 0;
  hipMalloc(&d_in, sizeof(float) * length);
  hipMalloc(&d_out, sizeof(float));
  hipMemcpyAsync(d_in, arr, sizeof(float) * length, hipMemcpyDefault);
  hipcub::DeviceReduce::Sum(d_temp, temp_bytes, d_in, d_out, length);
  hipMalloc(&d_temp, temp_bytes);
  hipcub::DeviceReduce::Sum(d_temp, temp_bytes, d_in, d_out, length);
  float out;
  hipMemcpy(&out, d_out, sizeof(float), hipMemcpyDefault);
  hipFree(d_temp);
  hipFree(d_in);
  hipFree(d_out);
  return out;
}